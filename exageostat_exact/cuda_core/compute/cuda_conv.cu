#include "hip/hip_runtime.h"
/**
 *
 * Copyright (c) 2017-2023  King Abdullah University of Science and Technology
 * All rights reserved.
 *
 * ExaGeoStat is a software package provided by KAUST
 **/
/**
 *
 * @file cudaconv.c
 *
 * Cuda datatypes conversion.
 *
 * @version 1.2.0
 *
 * @author Sameh Abdulah
 * @date 2022-11-09
 *
 **/

#define CHUNKSIZE 32

#include <hipblas.h>
#include <stdio.h>
#include "../include/exageostatcudacore.h"

__global__ void
float2double_array_kernel(int nrows, int ncols, const float *F, int ldf, double* H, int ldh, hipblasOperation_t transa) {
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int idx = blockIdx.x * blockDim.x + tx;
    const int idy = blockIdx.y * blockDim.y + ty;

    if (idx >= nrows || idy >= ncols) {
        return;
    }
    if (transa == HIPBLAS_OP_N)
        H[idy * ldh + idx] = (double) F[idy * ldf + idx];
    else
        H[idx * ldh + idy] = (double) F[idy * ldf + idx];
}

void float2double_array(int nrows, int ncols, const float *F, int ldf, double* H, int ldh, hipblasOperation_t transa,
                        hipStream_t stream) {
    int nBlockx = (nrows + CHUNKSIZE - 1) / CHUNKSIZE;
    int nBlocky = (ncols + CHUNKSIZE - 1) / CHUNKSIZE;
    dim3 dimBlock(CHUNKSIZE, CHUNKSIZE);
    dim3 dimGrid(nBlockx, nBlocky);
    float2double_array_kernel<<<dimGrid, dimBlock, 0, stream>>>(nrows, ncols, F, ldf, H, ldh, transa);
}

__global__ void
double2float_array_kernel(int nrows, int ncols, const double* H, int ldh, float *F, int ldf, hipblasOperation_t transa) {
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int idx = blockIdx.x * blockDim.x + tx;
    const int idy = blockIdx.y * blockDim.y + ty;

    if (idx >= nrows || idy >= ncols) {
        return;
    }

    if (transa == HIPBLAS_OP_N)
        F[idy * ldf + idx] = __double2float_rn(
                H[idy * ldh + idx]); //Convert a double to a float in round-to-nearest-even mode.
    else
        F[idx * ldf + idy] = __double2float_rn(
                H[idy * ldh + idx]); //Convert a double to a float in round-to-nearest-even mode
}

void double2float_array(int nrows, int ncols, const double* H, int ldh, float *F, int ldf, hipblasOperation_t transa,
                        hipStream_t stream) {

    int nBlockx = (nrows + CHUNKSIZE - 1) / CHUNKSIZE;
    int nBlocky = (ncols + CHUNKSIZE - 1) / CHUNKSIZE;
    dim3 dimBlock(CHUNKSIZE, CHUNKSIZE);
    dim3 dimGrid(nBlockx, nBlocky);
    double2float_array_kernel<<<dimGrid, dimBlock, 0, stream>>>(nrows, ncols, H, ldh, F, ldf, transa);
}